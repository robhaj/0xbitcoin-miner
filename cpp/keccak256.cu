#include "hip/hip_runtime.h"
/*
 * Keccak 256
 *
 */

#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_keccak.h"
#include "cuda.cu"
#include "cuda_keccak256.cu"
#include "keccak256.h"
#include <cstring>

static uint32_t h_nounce[MAX_GPUS][2];

// CPU Hash
void keccak256_hash(void *state, const void *input)
{
	printf("inside hashing method\n");
	sph_keccak_context ctx_keccak;

	uint32_t hash[16];

	printf("before sph_keccak256_init\n");
	sph_keccak256_init(&ctx_keccak);
	printf("before sph_keccak256\n");
	sph_keccak256 (&ctx_keccak, input, 80);
	printf("before sph_keccak256_close\n");
	sph_keccak256_close(&ctx_keccak, (void*) hash);

	printf("before memcpy\n");
	memcpy(state, hash, 32);
	printf("after memcpy\n");
}

static bool init[MAX_GPUS] = { 0 };

int scanhash_keccak256(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	printf("something\n");
	const uint32_t first_nonce = pdata[19];
	uint32_t intensity = (device_sm[device_map[thr_id]] > 500) ? 1 << 28 : 1 << 27;;
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity); // 256*4096
	throughput = min(throughput, max_nonce - first_nonce);

	printf ("intensity: %u throughput %u \n", intensity, throughput);
	printf ("first %i max %i \n", first_nonce, max_nonce);

	if (!init[thr_id]) {
		printf("init\n");
		hipSetDevice(device_map[thr_id]);
		//if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		//if (opt_n_gputhreads == 1)
		//{
		//	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		//}
		keccak256_cpu_init(thr_id, (int)throughput);
		init[thr_id] = true;
	}

	printf("post init\n");
	uint32_t endiandata[20];
	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
	}

	printf("before set block\n");
	keccak256_setBlock_80((void*)endiandata, (uint64_t *)ptarget);
	printf("after set block\n");
	do {

		keccak256_cpu_hash_80(thr_id, (int) throughput, pdata[19], h_nounce[thr_id]);
		printf("after keccak256_cpu_hash_80\n");
		if (h_nounce[thr_id][0] != UINT32_MAX)
		{
			printf("in the loop\n");
			uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], h_nounce[thr_id][0]);
			keccak256_hash(vhash64, endiandata);

			printf("in the loop 2\n");

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				printf("in the second if \n");
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (h_nounce[thr_id][1] != 0xffffffff)
				{
					pdata[21] = h_nounce[thr_id][1];
					res++;
				}
				pdata[19] = h_nounce[thr_id][0];
				return res;
			}
		}

		printf("after the loop\n");
		pdata[19] += throughput;
	} while (((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));
	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
