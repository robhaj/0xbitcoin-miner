#include "hip/hip_runtime.h"
/*
 * Keccak 256
 *
 */

#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_keccak.h"
#include "cuda.cu"
#include "cuda_keccak256.cu"
#include "keccak256.h"
#include <cstring>

static uint32_t h_nounce[MAX_GPUS][2];

// CPU Hash
void keccak256_hash(void *state, const void *input)
{
	sph_keccak_context ctx_keccak;

	uint32_t hash[16];

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256 (&ctx_keccak, input, 80);
	sph_keccak256_close(&ctx_keccak, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

int scanhash_keccak256(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t intensity = (device_sm[device_map[thr_id]] > 500) ? 1 << 28 : 1 << 27;;
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity); // 256*4096
	throughput = min(throughput, max_nonce - first_nonce);

//	printf ("intensity: %u throughput %u \n", intensity, throughput);
//	printf ("first %i max %i \n", first_nonce, max_nonce);

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);
		//if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		//if (opt_n_gputhreads == 1)
		//{
		//	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		//}
		keccak256_cpu_init(thr_id, (int)throughput);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
	}

	keccak256_setBlock_80((void*)endiandata, (uint64_t *)ptarget);
	do {

		keccak256_cpu_hash_80(thr_id, (int) throughput, pdata[19], h_nounce[thr_id]);
		if (h_nounce[thr_id][0] != UINT32_MAX)
		{
			uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], h_nounce[thr_id][0]);
			keccak256_hash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (h_nounce[thr_id][1] != 0xffffffff)
				{
					pdata[21] = h_nounce[thr_id][1];
					res++;
				}
				pdata[19] = h_nounce[thr_id][0];
				return res;
			}
		}

		pdata[19] += throughput;
	} while (((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));
	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
